#include "hip/hip_runtime.h"
#include "Utility.cuh"

#include <SahKdTree.hpp>

#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>
#include <thrust/zip_function.h>

#include <limits>

#include <cassert>

void SahKdTree::Projection::findPerfectSplit(const Params & sah, U layerSize, const thrust::device_vector<U> & layerNodeOffset, const Projection & y, const Projection & z)
{
    Timer timer;
    auto eventCount = U(event.kind.size());

    {
        event.polygonCountLeft.resize(eventCount);
        event.polygonCountRight.resize(eventCount);

        auto leftTriangleCountBegin = thrust::make_transform_iterator(event.kind.cbegin(), [] __host__ __device__(I eventKind) -> U { return (eventKind < 0) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.cbegin(), event.node.cend(), leftTriangleCountBegin, event.polygonCountLeft.begin());

        auto rightTriangleCountBegin = thrust::make_transform_iterator(event.kind.crbegin(), [] __host__ __device__(I eventKind) -> U { return (0 < eventKind) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.crbegin(), event.node.crend(), rightTriangleCountBegin, event.polygonCountRight.rbegin());
    }
    timer(" findPerfectSplit 2 * exclusive_scan_by_key");  // 4.131ms

    layer.splitCost.resize(layerSize);
    layer.splitEvent.resize(layerSize);
    layer.splitPos.resize(layerSize);

    layer.polygonCountLeft.resize(layerSize);
    layer.polygonCountRight.resize(layerSize);
    timer(" findPerfectSplit resize");  // 0.640ms

    auto nodeLimitsBegin = thrust::make_zip_iterator(thrust::make_tuple(node.min.cbegin(), node.max.cbegin(), y.node.min.cbegin(), y.node.max.cbegin(), z.node.min.cbegin(), z.node.max.cbegin()));
    auto nodeBboxBegin = thrust::make_permutation_iterator(nodeLimitsBegin, event.node.cbegin());
    using NodeBboxType = IteratorValueType<decltype(nodeBboxBegin)>;
    auto splitEventBegin = thrust::make_counting_iterator<U>(0);
    auto perfectSplitInputBegin = thrust::make_zip_iterator(thrust::make_tuple(nodeBboxBegin, event.pos.cbegin(), event.kind.cbegin(), splitEventBegin, event.polygonCountLeft.cbegin(), event.polygonCountRight.cbegin()));
    auto perfectSplitBegin = thrust::make_zip_iterator(thrust::make_tuple(layer.splitCost.begin(), layer.splitEvent.begin(), layer.splitPos.begin(), layer.polygonCountLeft.begin(), layer.polygonCountRight.begin()));
    auto perfectSplitOutputBegin = thrust::make_permutation_iterator(perfectSplitBegin, layerNodeOffset.cbegin());
    using PerfectSplitType = IteratorValueType<decltype(perfectSplitOutputBegin)>;
    auto toPerfectSplit = [sah] __host__ __device__(NodeBboxType nodeBbox, F splitPos, I eventKind, U splitEvent, U polygonCountLeft, U polygonCountRight) -> PerfectSplitType {
        F min = thrust::get<0>(nodeBbox), max = thrust::get<1>(nodeBbox);
        if (!(min < max)) {
            return {std::numeric_limits<F>::infinity(), splitEvent, splitPos, polygonCountLeft, polygonCountRight};
        }
        F l = splitPos - min;
        F r = max - splitPos;
        if (eventKind < 0) {
            ++splitEvent;
        } else if (eventKind == 0) {
            if ((l < r) ? (polygonCountLeft != 0) : (polygonCountRight == 0)) {
                ++polygonCountLeft;
                ++splitEvent;
            } else {
                ++polygonCountRight;
            }
        }
        F x = max - min;
        F y = thrust::get<3>(nodeBbox) - thrust::get<2>(nodeBbox);
        F z = thrust::get<5>(nodeBbox) - thrust::get<4>(nodeBbox);
        F perimeter = y + z;
        F area = y * z;
        if (!(F(0) < perimeter)) {
            assert(!(F(0) < area));
            perimeter = F(1);
        }
        F splitCost = (polygonCountLeft * (area + perimeter * l) + polygonCountRight * (area + perimeter * r)) / (area + perimeter * x);
        splitCost *= sah.intersectionCost;
        splitCost += sah.traversalCost;
        if ((polygonCountLeft == 0) || (polygonCountRight == 0)) {
            if ((min < splitPos) && (splitPos < max)) {
                splitCost *= sah.emptinessFactor;
            }
        }
        return {splitCost, splitEvent, splitPos, polygonCountLeft, polygonCountRight};
    };
    auto perfectSplitValueBegin = thrust::make_transform_iterator(perfectSplitInputBegin, thrust::zip_function(toPerfectSplit));
    [[maybe_unused]] auto ends = thrust::reduce_by_key(event.node.cbegin(), event.node.cend(), perfectSplitValueBegin, thrust::make_discard_iterator(), perfectSplitOutputBegin, thrust::equal_to<U>{}, thrust::minimum<PerfectSplitType>{});
    assert(ends.first == thrust::make_discard_iterator(layerNodeOffset.size()));
    timer(" findPerfectSplit reduce_by_key");  // 2.897ms
}
