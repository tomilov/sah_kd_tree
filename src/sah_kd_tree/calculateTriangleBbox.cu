#include "hip/hip_runtime.h"
#include "sah_kd_tree.cuh"

#include <thrust/advance.h>
#include <thrust/extrema.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <cassert>

namespace SahKdTree
{
template<I dimension>
void Projection<dimension>::calculateTriangleBbox()
{
    auto triangleCount = U(triangle.a.size());
    assert(triangleCount == U(triangle.b.size()));
    assert(triangleCount == U(triangle.c.size()));

    polygon.min.resize(triangleCount);
    polygon.max.resize(triangleCount);

    auto triangleBegin = thrust::make_zip_iterator(thrust::make_tuple(triangle.a.cbegin(), triangle.b.cbegin(), triangle.c.cbegin()));
    using TriangleType = IteratorValueType<decltype(triangleBegin)>;
    auto polygonBboxBegin = thrust::make_zip_iterator(thrust::make_tuple(polygon.min.begin(), polygon.max.begin()));
    using PolygonBboxType = IteratorValueType<decltype(polygonBboxBegin)>;
    auto toTriangleBbox = [] __host__ __device__(TriangleType triangle) -> PolygonBboxType {
        F a = thrust::get<0>(triangle);
        F b = thrust::get<1>(triangle);
        F c = thrust::get<2>(triangle);
        return {thrust::min(a, thrust::min(b, c)), thrust::max(a, thrust::max(b, c))};
    };
    thrust::transform(triangleBegin, thrust::next(triangleBegin, triangleCount), polygonBboxBegin, toTriangleBbox);
}

template void Projection<0>::calculateTriangleBbox();
template void Projection<1>::calculateTriangleBbox();
template void Projection<2>::calculateTriangleBbox();
}  // namespace SahKdTree
