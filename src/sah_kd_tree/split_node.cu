#include "hip/hip_runtime.h"
#include <sah_kd_tree/sah_kd_tree.cuh>

#include <thrust/advance.h>
#include <thrust/scatter.h>

void sah_kd_tree::Projection::splitNode(I dimension, U layerBasePrev, U layerBase, const thrust::device_vector<I> & nodeSplitDimension, const thrust::device_vector<F> & nodeSplitPos, const thrust::device_vector<U> & nodeLeft,
                                        const thrust::device_vector<U> & nodeRight)
{
    auto nodeSplitPosBegin = thrust::next(nodeSplitPos.cbegin(), layerBasePrev);
    auto nodeSplitPosEnd = thrust::next(nodeSplitPos.cbegin(), layerBase);
    auto nodeSplitDimensionBegin = thrust::next(nodeSplitDimension.cbegin(), layerBasePrev);
    const auto isX = [dimension] __host__ __device__(I nodeSplitDimension) -> bool { return nodeSplitDimension == dimension; };
    thrust::scatter_if(nodeSplitPosBegin, nodeSplitPosEnd, thrust::next(nodeLeft.cbegin(), layerBasePrev), nodeSplitDimensionBegin, node.max.begin(), isX);
    thrust::scatter_if(nodeSplitPosBegin, nodeSplitPosEnd, thrust::next(nodeRight.cbegin(), layerBasePrev), nodeSplitDimensionBegin, node.min.begin(), isX);
}
