#include "hip/hip_runtime.h"
#include <sah_kd_tree/sah_kd_tree.cuh>

#include <thrust/advance.h>
#include <thrust/scatter.h>

namespace sah_kd_tree
{
template<I dimension>
void Builder::splitNode(U layerBasePrev, Projection & projection)
{
    auto nodeSplitPosBegin = thrust::next(node.splitPos.cbegin(), layerBasePrev);
    auto nodeSplitPosEnd = thrust::next(node.splitPos.cbegin(), layer.base);
    auto nodeSplitDimensionBegin = thrust::next(node.splitDimension.cbegin(), layerBasePrev);
    const auto isCurrentProjection = [] __host__ __device__(I nodeSplitDimension) -> bool { return nodeSplitDimension == dimension; };
    thrust::scatter_if(nodeSplitPosBegin, nodeSplitPosEnd, thrust::next(node.leftChild.cbegin(), layerBasePrev), nodeSplitDimensionBegin, projection.node.max.begin(), isCurrentProjection);
    thrust::scatter_if(nodeSplitPosBegin, nodeSplitPosEnd, thrust::next(node.rightChild.cbegin(), layerBasePrev), nodeSplitDimensionBegin, projection.node.min.begin(), isCurrentProjection);
}

template void Builder::splitNode<0>(U layerBasePrev, Projection & x);
template void Builder::splitNode<1>(U layerBasePrev, Projection & y);
template void Builder::splitNode<2>(U layerBasePrev, Projection & z);
}  // namespace sah_kd_tree
