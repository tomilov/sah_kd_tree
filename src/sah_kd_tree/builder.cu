#include "hip/hip_runtime.h"
#include <sah_kd_tree/sah_kd_tree.cuh>

#include <thrust/advance.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scatter.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>

#include <cassert>

auto sah_kd_tree::Builder::operator()(const Params & sah, Projection & x, Projection & y, Projection & z) -> Tree
{
    x.calculateTriangleBbox();
    y.calculateTriangleBbox();
    z.calculateTriangleBbox();

    x.calculateRootNodeBbox();
    y.calculateRootNodeBbox();
    z.calculateRootNodeBbox();

    x.generateInitialEvent();
    y.generateInitialEvent();
    z.generateInitialEvent();

    polygon.triangle.resize(polygon.count);
    thrust::sequence(polygon.triangle.begin(), polygon.triangle.end());
    polygon.node.assign(polygon.count, U(0));

    node.splitDimension.resize(1);
    node.splitPos.resize(1);
    node.leftChild.resize(1);
    node.rightChild.resize(1);
    node.polygonCount.assign(1, polygon.count);
    node.polygonCountLeft.resize(1);
    node.polygonCountRight.resize(1);

    Tree tree;
    for (tree.depth = 0; tree.depth < sah.maxDepth; ++tree.depth) {
        filterLayerNodeOffset();

        x.findPerfectSplit(sah, layer.size, layer.nodeOffset, node.polygonCount, y, z);
        y.findPerfectSplit(sah, layer.size, layer.nodeOffset, node.polygonCount, z, x);
        z.findPerfectSplit(sah, layer.size, layer.nodeOffset, node.polygonCount, x, y);

        selectNodeBestSplit(sah, x, y, z);

        auto layerSplitDimensionBegin = thrust::next(node.splitDimension.cbegin(), layer.base);
        auto layerSplitDimensionEnd = thrust::next(layerSplitDimensionBegin, layer.size);
        auto layerLeafNodeCount = U(thrust::count(layerSplitDimensionBegin, layerSplitDimensionEnd, I(-1)));
        node.leafCount += layerLeafNodeCount;
        if (layerLeafNodeCount == layer.size) {
            break;
        }

        polygon.side.resize(polygon.count);
        polygon.eventRight.resize(polygon.count);

        determinePolygonSide<0>(x);
        determinePolygonSide<1>(y);
        determinePolygonSide<2>(z);

        updateSplittedPolygonCount();

        {  // generate index for child node
            auto nodeLeftChildBegin = thrust::next(node.leftChild.begin(), layer.base);
            const auto toNodeCount = [] __host__ __device__(I layerSplitDimension) -> U { return (layerSplitDimension < 0) ? 0 : 2; };
            auto nodeLeftChildEnd = thrust::transform_exclusive_scan(layerSplitDimensionBegin, layerSplitDimensionEnd, nodeLeftChildBegin, toNodeCount, layer.base + layer.size, thrust::plus<U>{});

            auto nodeRightChildBegin = thrust::next(node.rightChild.begin(), layer.base);
            const auto toNodeRightChild = [] __host__ __device__(U nodeLeftChild) -> U { return nodeLeftChild + 1; };
            thrust::transform(nodeLeftChildBegin, nodeLeftChildEnd, nodeRightChildBegin, toNodeRightChild);
        }

        separateSplittedPolygon();

        x.decoupleEventBoth(node.splitDimension, polygon.side);
        y.decoupleEventBoth(node.splitDimension, polygon.side);
        z.decoupleEventBoth(node.splitDimension, polygon.side);

        assert(polygon.side.size() == polygon.count);
        updatePolygonNode();

        splitPolygon<0>(x, y, z);
        splitPolygon<1>(y, z, x);
        splitPolygon<2>(z, x, y);

        updateSplittedPolygonNode();

        x.mergeEvent(polygon.count, polygon.splittedCount, polygon.node, splittedPolygon);
        y.mergeEvent(polygon.count, polygon.splittedCount, polygon.node, splittedPolygon);
        z.mergeEvent(polygon.count, polygon.splittedCount, polygon.node, splittedPolygon);

        U layerBasePrev = layer.base;
        layer.base += layer.size;
        layer.size -= layerLeafNodeCount;
        layer.size += layer.size;

        node.count = layer.base + layer.size;

        node.polygonCount.resize(node.count);

        auto nodePolygonCountLeftBegin = thrust::next(node.polygonCountLeft.cbegin(), layerBasePrev);
        auto nodePolygonCountLeftEnd = thrust::next(node.polygonCountLeft.cbegin(), layer.base);
        thrust::scatter_if(nodePolygonCountLeftBegin, nodePolygonCountLeftEnd, thrust::next(node.leftChild.cbegin(), layerBasePrev), layerSplitDimensionBegin, node.polygonCount.begin(), IsNotLeaf{});
        auto nodePolygonCountRightBegin = thrust::next(node.polygonCountRight.cbegin(), layerBasePrev);
        auto nodePolygonCountRightEnd = thrust::next(node.polygonCountRight.cbegin(), layer.base);
        thrust::scatter_if(nodePolygonCountRightBegin, nodePolygonCountRightEnd, thrust::next(node.rightChild.cbegin(), layerBasePrev), layerSplitDimensionBegin, node.polygonCount.begin(), IsNotLeaf{});

        setNodeCount(x, y, z);

        auto nodeBboxBegin = thrust::make_zip_iterator(x.node.min.begin(), x.node.max.begin(), y.node.min.begin(), y.node.max.begin(), z.node.min.begin(), z.node.max.begin());
        auto layerBboxBegin = thrust::next(nodeBboxBegin, layerBasePrev);
        auto layerBboxEnd = thrust::next(nodeBboxBegin, layer.base);
        thrust::scatter_if(layerBboxBegin, layerBboxEnd, thrust::next(node.leftChild.cbegin(), layerBasePrev), layerSplitDimensionBegin, nodeBboxBegin, IsNotLeaf{});
        thrust::scatter_if(layerBboxBegin, layerBboxEnd, thrust::next(node.rightChild.cbegin(), layerBasePrev), layerSplitDimensionBegin, nodeBboxBegin, IsNotLeaf{});

        splitNode<0>(layerBasePrev, x);
        splitNode<1>(layerBasePrev, y);
        splitNode<2>(layerBasePrev, z);

        node.splitDimension.resize(node.count, I(-1));
        node.splitPos.resize(node.count);
        node.leftChild.resize(node.count);
        node.rightChild.resize(node.count);
        node.polygonCountLeft.resize(node.count);
        node.polygonCountRight.resize(node.count);

        polygon.count += polygon.splittedCount;
    }

    node.parent.resize(node.count);
    thrust::scatter_if(thrust::make_counting_iterator<U>(0), thrust::make_counting_iterator<U>(node.count), node.leftChild.cbegin(), node.splitDimension.cbegin(), node.parent.begin(), IsNotLeaf{});
    thrust::scatter_if(thrust::make_counting_iterator<U>(0), thrust::make_counting_iterator<U>(node.count), node.rightChild.cbegin(), node.splitDimension.cbegin(), node.parent.begin(), IsNotLeaf{});

    assert(checkTree(x, y, z));

    populateLeafNodeTriangleRange();

    calculateRope<0>(Direction::kNegative, x, y, z);
    calculateRope<0>(Direction::kPositive, x, y, z);

    calculateRope<1>(Direction::kNegative, y, z, x);
    calculateRope<1>(Direction::kPositive, y, z, x);

    calculateRope<2>(Direction::kNegative, z, x, y);
    calculateRope<2>(Direction::kPositive, z, x, y);

    return tree;
}
