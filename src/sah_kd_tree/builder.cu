#include "hip/hip_runtime.h"
#include "sah_kd_tree/sah_kd_tree.hpp"
#include "sah_kd_tree/utility.cuh"

#include <thrust/advance.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scatter.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/tuple.h>

#if 0
#include <thrust/host_vector.h>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_reference.h>

#include <thrust/system/cuda/vector.h>
#include <thrust/system/cuda/pointer.h>

#include <thrust/system/cpp/vector.h>
#include <thrust/system/cpp/pointer.h>

#include <thrust/system/omp/vector.h>
#include <thrust/system/omp/pointer.h>

#include <thrust/system/tbb/vector.h>
#include <thrust/system/tbb/pointer.h>

#include <thrust/pair.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/transform_input_output_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/complex.h>
#include <iostream>
#endif

#include <cassert>

auto sah_kd_tree::Builder::operator()(const Params & sah) -> Tree
{
#if 0
    {
        auto pppp = thrust::make_pair(1, 2);
        auto cccc = thrust::complex<int>(1, 2);
        {
            auto ppp = thrust::make_constant_iterator(123);
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_counting_iterator(123);
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_discard_iterator(123);
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_reverse_iterator(thrust::make_constant_iterator(123));
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_transform_input_output_iterator(thrust::make_counting_iterator(123), [](int i) { return i; }, [](int j) { return j; });
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_transform_iterator(thrust::make_counting_iterator(123), [](int i) { return i; });
            std::cout << (void*)&ppp << std::endl;
        }
        {
            auto ppp = thrust::make_transform_output_iterator(thrust::make_counting_iterator(123), [](int i) { return i; });
            std::cout << (void*)&ppp << std::endl;
        }
        {
            thrust::host_vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            decltype(auto) ppp = v.data();
            decltype(auto) ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
        {
            thrust::device_vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            thrust::device_ptr<int> ppp = v.data();
            thrust::device_reference<int> ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
        {
            thrust::system::cuda::vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            thrust::system::cuda::pointer<int> ppp = v.data();
            thrust::system::cuda::reference<int> ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
        {
            thrust::system::cpp::vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            thrust::system::cpp::pointer<int> ppp = v.data();
            auto ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
        {
            thrust::system::omp::vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            thrust::system::omp::pointer<int> ppp = v.data();
            thrust::system::omp::reference<int> ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
        {
            thrust::system::tbb::vector<int> v(5);
            thrust::sequence(v.rbegin(), v.rend());
            thrust::system::tbb::pointer<int> ppp = v.data();
            thrust::system::tbb::reference<int> ref = v.front();
            thrust::sequence(v.rbegin(), v.rend());
        }
    }
#endif

    Timer timerTotal;
    Timer timer;

    auto triangleCount = U(x.triangle.a.size());
    assert(triangleCount == U(y.triangle.a.size()));
    assert(triangleCount == U(z.triangle.a.size()));

    if (triangleCount == 0) {
        return {};
    }

    x.calculateTriangleBbox();
    y.calculateTriangleBbox();
    z.calculateTriangleBbox();
    timer("calculateTriangleBbox");  // 9.330ms

    x.calculateRootNodeBbox();
    y.calculateRootNodeBbox();
    z.calculateRootNodeBbox();
    timer("calculateRootNodeBbox");  // 2.358ms

    x.generateInitialEvent();
    y.generateInitialEvent();
    z.generateInitialEvent();
    timer("generateInitialEvent");  // 141.887ms

    polygon.triangle.resize(triangleCount);
    thrust::sequence(polygon.triangle.begin(), polygon.triangle.end());
    polygon.node.assign(triangleCount, U(0));

    node.splitDimension.resize(1);
    node.splitPos.resize(1);
    node.nodeLeft.resize(1);
    node.nodeRight.resize(1);
    node.polygonCount.assign(1, triangleCount);
    node.polygonCountLeft.resize(1);
    node.polygonCountRight.resize(1);
    timer("init builder");  // 2.359ms

    // layer
    U layerBase = 0;
    U layerSize = 1;

    Tree tree;
    for (tree.depth = 0; tree.depth < sah.maxDepth; ++tree.depth) {
        thinLayerNodeOffset(layerBase, layerSize);
        timer("layerNodeOffset");  // 0.074ms

        x.findPerfectSplit(sah, layerSize, layerNodeOffset, y, z);
        y.findPerfectSplit(sah, layerSize, layerNodeOffset, z, x);
        z.findPerfectSplit(sah, layerSize, layerNodeOffset, x, y);
        timer("findPerfectSplit");  // 20.758ms

        selectNodeBestSplit(sah, layerBase, layerSize);
        timer("selectNodeBestSplit");  // 0.202ms

        auto layerSplitDimensionBegin = thrust::next(node.splitDimension.cbegin(), layerBase);
        auto layerSplitDimensionEnd = thrust::next(layerSplitDimensionBegin, layerSize);
        auto completedNodeCount = U(thrust::count(layerSplitDimensionBegin, layerSplitDimensionEnd, I(-1)));
        timer("completedNodeCount");  // 0.256ms
        if (completedNodeCount == layerSize) {
            break;
        }

        auto polygonCount = U(polygon.triangle.size());

        polygon.side.resize(polygonCount);
        polygon.eventRight.resize(polygonCount);
        timer("resize polygon");  // 0.944ms

        x.determinePolygonSide(0, node.splitDimension, layerBase, polygon.eventRight, polygon.side);
        y.determinePolygonSide(1, node.splitDimension, layerBase, polygon.eventRight, polygon.side);
        z.determinePolygonSide(2, node.splitDimension, layerBase, polygon.eventRight, polygon.side);
        timer("determinePolygonSide");  // 7.020ms

        U splittedPolygonCount = getSplittedPolygonCount(layerBase, layerSize);
        timer("getSplittedPolygonCount");  // 0.048ms

        {  // generate index for child node
            auto nodeLeftBegin = thrust::next(node.nodeLeft.begin(), layerBase);
            auto toNodeCount = [] __host__ __device__(I layerSplitDimension) -> U { return (layerSplitDimension < 0) ? 0 : 2; };
            auto nodeLeftEnd = thrust::transform_exclusive_scan(layerSplitDimensionBegin, layerSplitDimensionEnd, nodeLeftBegin, toNodeCount, layerBase + layerSize, thrust::plus<U>{});

            auto nodeRightBegin = thrust::next(node.nodeRight.begin(), layerBase);
            auto toNodeRight = [] __host__ __device__(U nodeLeft) { return nodeLeft + 1; };
            thrust::transform(nodeLeftBegin, nodeLeftEnd, nodeRightBegin, toNodeRight);
        }
        timer("toNodePairIndices");  // 0.052ms

        separateSplittedPolygon(layerBase, polygonCount, splittedPolygonCount);
        timer("separateSplittedPolygon");  // 0.516ms

        x.decoupleEventBoth(node.splitDimension, polygon.side);
        y.decoupleEventBoth(node.splitDimension, polygon.side);
        z.decoupleEventBoth(node.splitDimension, polygon.side);
        timer("decoupleEventBoth");  // 7.316ms

        assert(polygon.side.size() == polygonCount);
        updatePolygonNode(layerBase);
        timer("updatePolygonNode");  // 0.727ms

        x.splitPolygon(0, node.splitDimension, node.splitPos, polygon.triangle, polygon.node, polygonCount, splittedPolygonCount, splittedPolygon, y, z);
        y.splitPolygon(1, node.splitDimension, node.splitPos, polygon.triangle, polygon.node, polygonCount, splittedPolygonCount, splittedPolygon, z, x);
        z.splitPolygon(2, node.splitDimension, node.splitPos, polygon.triangle, polygon.node, polygonCount, splittedPolygonCount, splittedPolygon, x, y);
        timer("splitPolygon");  // 0.006ms

        updateSplittedPolygonNode(polygonCount, splittedPolygonCount);
        timer("updateSplittedPolygonNode");  // 0.003ms

        x.mergeEvent(polygonCount, splittedPolygonCount, polygon.node, splittedPolygon);
        y.mergeEvent(polygonCount, splittedPolygonCount, polygon.node, splittedPolygon);
        z.mergeEvent(polygonCount, splittedPolygonCount, polygon.node, splittedPolygon);
        timer("mergeEvent");  // 44.897ms

        U layerBasePrev = layerBase;
        layerBase += layerSize;
        layerSize -= completedNodeCount;
        layerSize += layerSize;

        node.polygonCount.resize(layerBase + layerSize);

        auto isNotLeaf = [] __host__ __device__(I layerSplitDimension) -> bool { return !(layerSplitDimension < 0); };

        auto nodePolygonCountLeftBegin = thrust::next(node.polygonCountLeft.cbegin(), layerBasePrev);
        auto nodePolygonCountLeftEnd = thrust::next(node.polygonCountLeft.cbegin(), layerBase);
        thrust::scatter_if(nodePolygonCountLeftBegin, nodePolygonCountLeftEnd, thrust::next(node.nodeLeft.cbegin(), layerBasePrev), layerSplitDimensionBegin, node.polygonCount.begin(), isNotLeaf);
        auto nodePolygonCountRightBegin = thrust::next(node.polygonCountRight.cbegin(), layerBasePrev);
        auto nodePolygonCountRightEnd = thrust::next(node.polygonCountRight.cbegin(), layerBase);
        thrust::scatter_if(nodePolygonCountRightBegin, nodePolygonCountRightEnd, thrust::next(node.nodeRight.cbegin(), layerBasePrev), layerSplitDimensionBegin, node.polygonCount.begin(), isNotLeaf);
        timer("polygonCount");  // 0.056ms

        x.setNodeCount(layerBase + layerSize);
        y.setNodeCount(layerBase + layerSize);
        z.setNodeCount(layerBase + layerSize);
        timer("setNodeCount");  // 0.174ms

        auto nodeBboxBegin = thrust::make_zip_iterator(thrust::make_tuple(x.node.min.begin(), x.node.max.begin(), y.node.min.begin(), y.node.max.begin(), z.node.min.begin(), z.node.max.begin()));
        auto layerBboxBegin = thrust::next(nodeBboxBegin, layerBasePrev);
        auto layerBboxEnd = thrust::next(nodeBboxBegin, layerBase);
        thrust::scatter_if(layerBboxBegin, layerBboxEnd, thrust::next(node.nodeLeft.cbegin(), layerBasePrev), layerSplitDimensionBegin, nodeBboxBegin, isNotLeaf);
        thrust::scatter_if(layerBboxBegin, layerBboxEnd, thrust::next(node.nodeRight.cbegin(), layerBasePrev), layerSplitDimensionBegin, nodeBboxBegin, isNotLeaf);
        timer("setNodeBbox");  // 0.031ms

        x.splitNode(0, layerBasePrev, layerBase, node.splitDimension, node.splitPos, node.nodeLeft, node.nodeRight);
        y.splitNode(1, layerBasePrev, layerBase, node.splitDimension, node.splitPos, node.nodeLeft, node.nodeRight);
        z.splitNode(2, layerBasePrev, layerBase, node.splitDimension, node.splitPos, node.nodeLeft, node.nodeRight);
        timer("splitNode");  // 0.062ms

        node.splitDimension.resize(layerBase + layerSize, I(-1));
        node.splitPos.resize(layerBase + layerSize);
        node.nodeLeft.resize(layerBase + layerSize);
        node.nodeRight.resize(layerBase + layerSize);
        node.polygonCountLeft.resize(layerBase + layerSize);
        node.polygonCountRight.resize(layerBase + layerSize);
        timer("resizeNode");  // 0.168ms
    }
    timerTotal("total");  // 236.149ms

    // calculate node parent
    // sort value (polygon) by key (polygon.node)
    // reduce value (counter, 1) by operation (project1st, plus) and key (node) to (key (node), value (offset, count))
    // scatter value (offset, count) to (node.nodeLeft, node.nodeRight) at key (node)
    return tree;
}
