#include "hip/hip_runtime.h"
#include <sah_kd_tree/sah_kd_tree.cuh>
#include <sah_kd_tree/utility.cuh>

#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>
#include <thrust/zip_function.h>

#include <limits>

#include <cassert>

void sah_kd_tree::Projection::findPerfectSplit(const Params & sah, U layerSize, const thrust::device_vector<U> & layerNodeOffset, const thrust::device_vector<U> & nodePolygonCount, const Projection & y, const Projection & z)
{
    Timer timer;
    auto eventCount = U(event.kind.size());

    {
        event.polygonCountLeft.resize(eventCount);
        event.polygonCountRight.resize(eventCount);

        auto leftTriangleCountBegin = thrust::make_transform_iterator(event.kind.cbegin(), [] __host__ __device__(I eventKind) -> U { return (eventKind < 0) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.cbegin(), event.node.cend(), leftTriangleCountBegin, event.polygonCountLeft.begin());

        auto rightTriangleCountBegin = thrust::make_transform_iterator(event.kind.crbegin(), [] __host__ __device__(I eventKind) -> U { return (0 < eventKind) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.crbegin(), event.node.crend(), rightTriangleCountBegin, event.polygonCountRight.rbegin());
    }
    timer(" findPerfectSplit 2 * exclusive_scan_by_key");  // 4.131ms

    layer.splitCost.resize(layerSize);
    layer.splitEvent.resize(layerSize);
    layer.splitPos.resize(layerSize);

    layer.polygonCountLeft.resize(layerSize);
    layer.polygonCountRight.resize(layerSize);
    timer(" findPerfectSplit resize");  // 0.640ms

    auto nodeLimitsBegin = thrust::make_zip_iterator(thrust::make_tuple(node.min.cbegin(), node.max.cbegin(), y.node.min.cbegin(), y.node.max.cbegin(), z.node.min.cbegin(), z.node.max.cbegin()));
    auto nodeBboxBegin = thrust::make_permutation_iterator(nodeLimitsBegin, event.node.cbegin());
    using NodeBboxType = IteratorValueType<decltype(nodeBboxBegin)>;
    auto splitEventBegin = thrust::make_counting_iterator<U>(0);
    auto polygonCount = thrust::make_permutation_iterator(nodePolygonCount.cbegin(), event.node.cbegin());
    auto perfectSplitInputBegin = thrust::make_zip_iterator(thrust::make_tuple(nodeBboxBegin, event.pos.cbegin(), event.kind.cbegin(), splitEventBegin, polygonCount, event.polygonCountLeft.cbegin(), event.polygonCountRight.cbegin()));
    auto perfectSplitBegin = thrust::make_zip_iterator(thrust::make_tuple(layer.splitCost.begin(), layer.splitEvent.begin(), layer.splitPos.begin(), layer.polygonCountLeft.begin(), layer.polygonCountRight.begin()));
    auto perfectSplitOutputBegin = thrust::make_permutation_iterator(perfectSplitBegin, layerNodeOffset.cbegin());
    using PerfectSplitType = IteratorValueType<decltype(perfectSplitOutputBegin)>;
    auto toPerfectSplit = [sah] __host__ __device__(NodeBboxType nodeBbox, F splitPos, I eventKind, U splitEvent, U polygonCount, U polygonCountLeft, U polygonCountRight) -> PerfectSplitType {
        F min = thrust::get<0>(nodeBbox), max = thrust::get<1>(nodeBbox);
        assert(!(splitPos < min));
        assert(!(max < splitPos));
        if (!(min < max)) {
            return {std::numeric_limits<F>::infinity()};
        }
        F l = splitPos - min;
        F r = max - splitPos;
        if (eventKind < 0) {
            assert(0 != polygonCountLeft);
            ++splitEvent;
        } else if (eventKind == 0) {
            if ((l < r) ? (polygonCountLeft != 0) : (polygonCountRight == 0)) {
                ++polygonCountLeft;
                ++splitEvent;
            } else {
                ++polygonCountRight;
            }
        } else {
            assert(0 != polygonCountRight);
        }
        if ((polygonCountLeft == polygonCount) || (polygonCountRight == polygonCount)) {
            return {std::numeric_limits<F>::infinity()};
        }
        F emptinessFactor(1);
        if (polygonCountLeft == 0) {
            assert(polygonCountRight != 0);
            if (!(min < splitPos)) {
                return {std::numeric_limits<F>::infinity()};
            }
            emptinessFactor = sah.emptinessFactor;
        } else if (polygonCountRight == 0) {
            if (!(splitPos < max)) {
                return {std::numeric_limits<F>::infinity()};
            }
            emptinessFactor = sah.emptinessFactor;
        }
        F x = max - min;
        F y = thrust::get<3>(nodeBbox) - thrust::get<2>(nodeBbox);
        F z = thrust::get<5>(nodeBbox) - thrust::get<4>(nodeBbox);
        F area = y * z;  // half area
        F splitCost;
        if (F(0) < area) {
            F perimeter = y + z;  // half perimeter
            assert(F(0) < perimeter);
            splitCost = (polygonCountLeft * (area + perimeter * l) + polygonCountRight * (area + perimeter * r)) / (area + perimeter * x);
        } else {
            splitCost = (polygonCountLeft * l + polygonCountRight * r) / x;
        }
        splitCost *= sah.intersectionCost;
        splitCost += sah.traversalCost;
        splitCost *= emptinessFactor;
        return {splitCost, splitEvent, splitPos, polygonCountLeft, polygonCountRight};
    };
    auto perfectSplitValueBegin = thrust::make_transform_iterator(perfectSplitInputBegin, thrust::zip_function(toPerfectSplit));
    [[maybe_unused]] auto ends = thrust::reduce_by_key(event.node.cbegin(), event.node.cend(), perfectSplitValueBegin, thrust::make_discard_iterator(), perfectSplitOutputBegin, thrust::equal_to<U>{}, thrust::minimum<PerfectSplitType>{});
    assert(ends.first == thrust::make_discard_iterator(layerNodeOffset.size()));
    timer(" findPerfectSplit reduce_by_key");  // 2.897ms
}
