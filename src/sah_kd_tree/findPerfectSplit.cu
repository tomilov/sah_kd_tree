#include "hip/hip_runtime.h"
#include "utility.cuh"

#include <sah_kd_tree/sah_kd_tree.hpp>

#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>
#include <thrust/zip_function.h>

#include <cassert>

void SahKdTree::Projection::findPerfectSplit(const Params & sah, U nodeCount, const thrust::device_vector<U> & layerNodeOffset, const Projection & y, const Projection & z)
{
    Timer timer;
    auto eventCount = U(event.kind.size());

    {
        event.polygonCountLeft.resize(eventCount);
        event.polygonCountRight.resize(eventCount);

        auto leftTriangleCountBegin = thrust::make_transform_iterator(event.kind.cbegin(), [] __host__ __device__(I eventKind) -> U { return (eventKind < 0) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.cbegin(), event.node.cend(), leftTriangleCountBegin, event.polygonCountLeft.begin());

        auto rightTriangleCountBegin = thrust::make_transform_iterator(event.kind.crbegin(), [] __host__ __device__(I eventKind) -> U { return (0 < eventKind) ? 0 : 1; });
        thrust::exclusive_scan_by_key(event.node.crbegin(), event.node.crend(), rightTriangleCountBegin, event.polygonCountRight.rbegin());
    }
    timer(" findPerfectSplit 2 * exclusive_scan_by_key");  // 4.131ms

    layer.splitCost.resize(nodeCount);
    layer.splitEvent.resize(nodeCount);
    layer.splitPos.resize(nodeCount);

    layer.polygonCountLeft.resize(nodeCount);
    layer.polygonCountRight.resize(nodeCount);
    timer(" findPerfectSplit resize");  // 0.640ms

    auto nodeLimitsBegin = thrust::make_zip_iterator(thrust::make_tuple(node.min.cbegin(), node.max.cbegin(), y.node.min.cbegin(), y.node.max.cbegin(), z.node.min.cbegin(), z.node.max.cbegin()));
    auto nodeBboxBegin = thrust::make_permutation_iterator(nodeLimitsBegin, event.node.cbegin());
    using NodeBboxType = IteratorValueType<decltype(nodeBboxBegin)>;
    auto splitEventBegin = thrust::make_counting_iterator<U>(0);
    auto perfectSplitInputBegin = thrust::make_zip_iterator(thrust::make_tuple(nodeBboxBegin, event.pos.cbegin(), event.kind.cbegin(), splitEventBegin, event.polygonCountLeft.cbegin(), event.polygonCountRight.cbegin()));
    auto perfectSplitBegin = thrust::make_zip_iterator(thrust::make_tuple(layer.splitCost.begin(), layer.splitEvent.begin(), layer.splitPos.begin(), layer.polygonCountLeft.begin(), layer.polygonCountRight.begin()));
    using PerfectSplitType = IteratorValueType<decltype(perfectSplitBegin)>;
    auto toPerfectSplit = [sah] __host__ __device__(NodeBboxType nodeBbox, F splitPos, I eventKind, U splitEvent, U polygonCountLeft, U polygonCountRight) -> PerfectSplitType {
        F min = thrust::get<0>(nodeBbox), max = thrust::get<1>(nodeBbox);
        F xLeft = splitPos - min;
        F xRight = max - splitPos;
        if (eventKind < 0) {
            ++splitEvent;
        } else if (eventKind == 0) {
            if ((xLeft < xRight) ? (polygonCountLeft != 0) : (polygonCountRight == 0)) {
                ++polygonCountLeft;
                ++splitEvent;
            } else {
                ++polygonCountRight;
            }
        }
        F x = max - min;
        F y = thrust::get<3>(nodeBbox) - thrust::get<2>(nodeBbox);
        F z = thrust::get<5>(nodeBbox) - thrust::get<4>(nodeBbox);
        F perimeter = y + z;
        F area = y * z;
        F splitCost = (polygonCountLeft * (area + perimeter * xLeft) + polygonCountRight * (area + perimeter * xRight)) / (area + perimeter * x);
        splitCost *= sah.intersectionCost;
        splitCost += sah.traversalCost;
        if ((polygonCountLeft == 0) || (polygonCountRight == 0)) {
            splitCost *= sah.emptinessFactor;
        }
        return {splitCost, splitEvent, splitPos, polygonCountLeft, polygonCountRight};
    };
    auto perfectSplitValueBegin = thrust::make_transform_iterator(perfectSplitInputBegin, thrust::zip_function(toPerfectSplit));
    [[maybe_unused]] auto ends = thrust::reduce_by_key(event.node.cbegin(), event.node.cend(), perfectSplitValueBegin, thrust::make_discard_iterator(), thrust::make_permutation_iterator(perfectSplitBegin, layerNodeOffset.cbegin()),
                                                       thrust::equal_to<U>{}, thrust::minimum<PerfectSplitType>{});
    assert(ends.first == thrust::make_discard_iterator(layerNodeOffset.size()));
    timer(" findPerfectSplit reduce_by_key");  // 2.897ms
}
