#include "hip/hip_runtime.h"
#include <sah_kd_tree/sah_kd_tree.cuh>
#include <sah_kd_tree/utility.cuh>

#include <thrust/advance.h>
#include <thrust/extrema.h>

void sah_kd_tree::Projection::calculateRootNodeBbox()
{
    auto rootBboxMinBegin = thrust::min_element(polygon.min.cbegin(), polygon.min.cend());
    node.min.assign(rootBboxMinBegin, thrust::next(rootBboxMinBegin));

    auto rootBboxMaxBegin = thrust::max_element(polygon.max.cbegin(), polygon.max.cend());
    node.max.assign(rootBboxMaxBegin, thrust::next(rootBboxMaxBegin));
}
